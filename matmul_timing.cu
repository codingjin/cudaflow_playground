// Core CUDA headers
#include <hip/hip_runtime.h>          // Mandatory
 // Recommended for kernel variables

#include <vector>
#include <random>
#include <iostream>
#include <cmath>
#include <algorithm>
#include <chrono>
#include <string>
#include <stdexcept> // For std::invalid_argument, std::out_of_range


// Matmul kernel
__global__ void matmul(const float* A, const float* B, float* C, int M, int N, int K) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    if (col < N && row < M) {
        float tmp = 0.0f;
        for (int i = 0; i < N; ++i) {
            tmp += A[row * N + i] * B[i * K + col];
        }
        C[row * K + col] = tmp;
    }
}


int main(int argc, char* argv[]) {
    if (argc != 2) {
        std::cerr << "Usage: " << argv[0] << " <integer>(>=1)" << std::endl;
        return 1; // Return error code
    }
    int IterationNum = 0;
    std::string arg = argv[1]; // Get the argument string
    try {
        size_t pos;
        int num = std::stoi(arg, &pos); // Convert to integer

        // Ensure entire string was processed (no extra characters)
        if (pos != arg.length()) {
            std::cerr << "Error: Argument must be a single integer." << std::endl;
            return 1;
        }

        if (num < 1) {
            std::cerr << "Error: Argument(integer) should be >= 1." << std::endl;
            return 1;
        }

        IterationNum = num;
    }
    catch (const std::invalid_argument&) {
        std::cerr << "Error: '" << arg << "' is not a valid integer." << std::endl;
        return 1;
    }
    catch (const std::out_of_range&) {
        std::cerr << "Error: '" << arg << "' is out of int range." << std::endl;
        return 1;
    }

    const unsigned N = 4096;
    const unsigned N2 = N*N;
    const unsigned RANDOM_SEED = 137;
    std::mt19937 gen(RANDOM_SEED);
    std::uniform_real_distribution<float> dist(0.0f, 1.0f);

    std::vector<float> ha, hb, hc;
    float* da{nullptr};
    float* db{nullptr};
    float* dc{nullptr};
    
    std::vector<float> result;
    ha.resize(N2);
    hb.resize(N2);
    hc.resize(N2, 0.0f);
    result.resize(N2);
    for (size_t i = 0; i < N2; ++i) {
        ha[i] = dist(gen);
        hb[i] = dist(gen);
    }

    dim3 grid((N + 31) / 32, (N + 31) / 32);
    dim3 block(32, 32);

    hipMalloc(&da, N2*sizeof(float));
    hipMalloc(&db, N2*sizeof(float));
    hipMalloc(&dc, N2*sizeof(float));

    hipStream_t stream;
    hipStreamCreate(&stream);
    
    auto gbeg = std::chrono::steady_clock::now();
    for (int i = 0; i < IterationNum; ++i) {
        hipMemcpyAsync(da, &ha[0], N2*sizeof(float), hipMemcpyHostToDevice, stream);
        hipMemcpyAsync(db, &hb[0], N2*sizeof(float), hipMemcpyHostToDevice, stream);
        matmul<<<grid, block, 0, stream>>>(da, db, dc, N, N, N);
        hipMemcpyAsync(&hc[0], dc, N2*sizeof(float), hipMemcpyDeviceToHost, stream);
    }
    hipStreamSynchronize(stream);
    auto gend = std::chrono::steady_clock::now();
    
    std::cout << "CUDA matmul M=N=K=" << N << " IterationNum=" << IterationNum << std::endl 
        << "completed with " 
        << std::chrono::duration_cast<std::chrono::milliseconds>(gend-gbeg).count()
        << " ms\n";
    
    hipFree(da);
    hipFree(db);
    hipFree(dc);
    hipStreamDestroy(stream);
    /*
    std::cout << "Verifying results of Matmul C = A*B" << std::endl;
    for (int i = 0; i < N; ++i) {
        for (int j = 0; j < N; ++j) {
            float tmp = 0.0f;
            for (int k = 0; k < N; ++k)
                tmp += ha[i * N + k] * hb[k * N + j];
            result[i * N + j] = tmp;
        }
    }

    float maxerr = 0.0f;
    for (size_t i = 0; i < N2; ++i) {
        maxerr = std::max(maxerr, std::abs(hc[i] - result[i]));
    }
    std::cout << "Max abs-error = " << maxerr << std::endl;
    */
    return 0;
}

