#include "hip/hip_runtime.h"
#include <taskflow/taskflow.hpp>
#include <taskflow/algorithm/for_each.hpp>
#include <taskflow/cuda/cudaflow.hpp>

#include <vector>
#include <random>
#include <iostream>
#include <cmath>
#include <algorithm>


// Matmul kernel
__global__ void matmul(const float* A, const float* B, float* C, int M, int N, int K) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    if (col < N && row < M) {
        float tmp = 0.0f;
        for (int i = 0; i < N; ++i) {
            tmp += A[row * N + i] * B[i * K + col];
        }
        C[row * K + col] = tmp;
    }
}

// Kernel: add
__global__ void add(const int n, const float *x, const float *y, float *z) {
    int idx = blockIdx.x*blockDim.x + threadIdx.x;
    if (idx < n) {
        z[idx] = x[idx] + y[idx];
    }
}

int main(int argc, char* argv[]) {
    if (argc != 2) {
        std::cerr << "Usage: " << argv[0] << " <integer>(>=1)" << std::endl;
        return 1; // Return error code
    }
    int IterationNum = 0;
    std::string arg = argv[1]; // Get the argument string
    try {
        size_t pos;
        int num = std::stoi(arg, &pos); // Convert to integer

        // Ensure entire string was processed (no extra characters)
        if (pos != arg.length()) {
            std::cerr << "Error: Argument must be a single integer." << std::endl;
            return 1;
        }

        if (num < 1) {
            std::cerr << "Error: Argument(integer) should be >= 1." << std::endl;
            return 1;
        }

        IterationNum = num;
    }
    catch (const std::invalid_argument&) {
        std::cerr << "Error: '" << arg << "' is not a valid integer." << std::endl;
        return 1;
    }
    catch (const std::out_of_range&) {
        std::cerr << "Error: '" << arg << "' is out of int range." << std::endl;
        return 1;
    }

    const unsigned N = 4096;
    const unsigned N2 = N*N;
    const unsigned RANDOM_SEED = 137;
    std::mt19937 gen(RANDOM_SEED);
    std::uniform_real_distribution<float> dist(0.0f, 1.0f);

    std::vector<float> ha, hb, hc;
    ha.resize(N2);
    hb.resize(N2);
    hc.resize(N2, 0.0f);
    for (size_t i = 0; i < N2; ++i) {
        ha[i] = dist(gen);
        hb[i] = dist(gen);
    }

    float* da{nullptr};
    float* db{nullptr};
    float* dc{nullptr};
    float* dab_add{nullptr};
    float* dab_mul{nullptr};
    
    dim3 grid((N + 31) / 32, (N + 31) / 32);
    dim3 block(32, 32);
    dim3 grid1((N2 + 1023) / 1024);
    dim3 block1(1024);
    hipMalloc(&da, N2*sizeof(float));
    hipMalloc(&db, N2*sizeof(float));
    hipMalloc(&dc, N2*sizeof(float));
    hipMalloc(&dab_add, N2*sizeof(float));
    hipMalloc(&dab_mul, N2*sizeof(float));

    tf::Taskflow taskflow("Add_Mul_Matmul");
    tf::Executor executor;

    auto cudaFlow = taskflow.emplace([&](){
        tf::cudaGraph cg;
        auto copy_da = cg.copy(da, ha.data(), N2);
        auto copy_db = cg.copy(db, hb.data(), N2);
        auto copy_hc = cg.copy(hc.data(), dc, N2);

        auto addkernel = cg.kernel(grid1, block1, 0, add, N2, da, db, dab_add);
        auto mulkernel = cg.kernel(grid, block, 0, matmul, da, db, dab_mul, N, N, N);
        auto matmulkernel = cg.kernel(grid, block, 0, matmul, dab_add, dab_mul, dc, N, N, N);

        addkernel.succeed(copy_da, copy_db)
                 .precede(matmulkernel);

        mulkernel.succeed(copy_da, copy_db)
                   .precede(matmulkernel);

        matmulkernel.precede(copy_hc);

        tf::cudaStream stream;
        tf::cudaGraphExec exec(cg);
        stream.run(exec).synchronize();
    }).name("cudaFlow");

    auto gbeg = std::chrono::steady_clock::now();
    for (int i = 0; i < IterationNum; ++i) {
        executor.run(taskflow).wait();
    }
    auto gend = std::chrono::steady_clock::now();

    std::cout << "CUDAFLOW add,mul,matmul M=N=K=" << N << " IterationNum=" << IterationNum << std::endl 
        << "completed with " 
        << std::chrono::duration_cast<std::chrono::milliseconds>(gend-gbeg).count()
        << " ms\n";

    hipFree(da);
    hipFree(db);
    hipFree(dc);
    hipFree(dab_add);
    hipFree(dab_mul);
    
    /*
    std::cout << "Verifying results of Matmul C = (A+B)*(A*B)" << std::endl;
    std::vector<float> hab_add, hab_mul;
    hab_add.resize(N2);
    hab_mul.resize(N2);
    for (size_t i = 0; i < N2; ++i) {
        hab_add[i] = ha[i] + hb[i];
    }
    for (int i = 0; i < N; ++i) {
        for (int j = 0; j < N; ++j) {
            float tmp = 0.0f;
            for (int k = 0; k < N; ++k)
                tmp += ha[i * N + k] * hb[k * N + j];
            hab_mul[i * N + j] = tmp;
        }
    }

    std::vector<float> result;
    result.resize(N2);
    for (int i = 0; i < N; ++i) {
        for (int j = 0; j < N; ++j) {
            float tmp = 0.0f;
            for (int k = 0; k < N; ++k)
                tmp += hab_add[i * N + k] * hab_mul[k * N + j];
            result[i * N + j] = tmp;
        }
    }
    float maxerr = 0.0f;
    for (size_t i = 0; i < N2; ++i) {
        maxerr = std::max(maxerr, std::abs(hc[i] - result[i]));
    }
    std::cout << "Max abs-error = " << maxerr << std::endl;
    */
    return 0;
}

